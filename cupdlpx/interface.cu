#include "hip/hip_runtime.h"
/*
Copyright 2025 Haihao Lu

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "interface.h"
#include "solver.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// helper function to allocate and fill or copy an array
static void fill_or_copy(double** dst, int n, const double* src, double fill_val) {
    *dst = (double*)safe_malloc((size_t)n * sizeof(double));
    if (src) memcpy(*dst, src, (size_t)n * sizeof(double));
    else     for (int i = 0; i < n; ++i) (*dst)[i] = fill_val;
}

// convert dense → CSR
static void dense_to_csr(const matrix_desc_t* desc,
                         int** row_ptr, int** col_ind, double** vals, int* nnz_out) {
    int m = desc->m, n = desc->n;
    double tol = (desc->zero_tolerance > 0) ? desc->zero_tolerance : 1e-12;

    // count nnz
    int nnz = 0;
    for (int i = 0; i < m * n; ++i) {
        if (fabs(desc->data.dense.A[i]) > tol) ++nnz;
    }

    *row_ptr = (int*)safe_malloc((size_t)(m + 1) * sizeof(int));
    *col_ind = (int*)safe_malloc((size_t)nnz * sizeof(int));
    *vals    = (double*)safe_malloc((size_t)nnz * sizeof(double));

    int nz = 0;
    for (int i = 0; i < m; ++i) {
        (*row_ptr)[i] = nz;
        for (int j = 0; j < n; ++j) {
            double v = desc->data.dense.A[i * n + j];
            if (fabs(v) > tol) {
                (*col_ind)[nz] = j;
                (*vals)[nz] = v;
                ++nz;
            }
        }
    }
    (*row_ptr)[m] = nz;
    *nnz_out = nz;
}

// create an lp_problem_t from a matrix
lp_problem_t* make_problem_from_matrix(
    const matrix_desc_t* A_desc,
    const double* objective_c,
    const double* objective_constant,
    const double* var_lb,
    const double* var_ub,
    const double* con_lb,
    const double* con_ub
) {
    lp_problem_t* prob = (lp_problem_t*)safe_malloc(sizeof(lp_problem_t));

    prob->num_variables   = A_desc->n;
    prob->num_constraints = A_desc->m;

    // handle matrix by format → convert to CSR if needed
    switch (A_desc->fmt) {
        case matrix_dense:
            dense_to_csr(A_desc,
                         &prob->constraint_matrix_row_pointers,
                         &prob->constraint_matrix_col_indices,
                         &prob->constraint_matrix_values,
                         &prob->constraint_matrix_num_nonzeros);
            break;

        case matrix_csr:
            prob->constraint_matrix_num_nonzeros = A_desc->data.csr.nnz;
            prob->constraint_matrix_row_pointers = (int*)safe_malloc((size_t)(A_desc->m + 1) * sizeof(int));
            prob->constraint_matrix_col_indices = (int*)safe_malloc((size_t)A_desc->data.csr.nnz * sizeof(int));
            prob->constraint_matrix_values = (double*)safe_malloc((size_t)A_desc->data.csr.nnz * sizeof(double));
            memcpy(prob->constraint_matrix_row_pointers, A_desc->data.csr.row_ptr, (size_t)(A_desc->m + 1) * sizeof(int));
            memcpy(prob->constraint_matrix_col_indices, A_desc->data.csr.col_ind, (size_t)A_desc->data.csr.nnz * sizeof(int));
            memcpy(prob->constraint_matrix_values, A_desc->data.csr.vals, (size_t)A_desc->data.csr.nnz * sizeof(double));
            break;

        // TODO: other formats
        default:
            fprintf(stderr, "[interface] make_problem_from_matrix: unsupported matrix format %d.\n", A_desc->fmt);
            free(prob);
            return NULL;
    }

    // default fill values
    prob->objective_constant = objective_constant ? *objective_constant : 0.0;
    fill_or_copy(&prob->objective_vector, prob->num_variables, objective_c, 0.0);
    fill_or_copy(&prob->variable_lower_bound, prob->num_variables, var_lb, 0.0);
    fill_or_copy(&prob->variable_upper_bound, prob->num_variables, var_ub, INFINITY);
    fill_or_copy(&prob->constraint_lower_bound, prob->num_constraints, con_lb, -INFINITY);
    fill_or_copy(&prob->constraint_upper_bound, prob->num_constraints, con_ub, INFINITY);

    return prob;
}

lp_solution_t solve_lp_problem(
    const lp_problem_t* prob,
    const pdhg_parameters_t* params
) {
    // initialize output
    lp_solution_t out = {0};

    // argument checks
    if (!prob) {
        fprintf(stderr, "[interface] solve_lp_problem: invalid arguments.\n");
        return out;
    }

    // prepare parameters: use defaults if not provided 
    pdhg_parameters_t local_params;
    if (params) {
        local_params = *params;
    } else {
        set_default_parameters(&local_params);
    }

    // call optimizer
    pdhg_solver_state_t* state = optimize(&local_params, prob);
    if (!state) {
        fprintf(stderr, "[interface] optimize returned NULL.\n");
        return out;
    }

    // prepare output
    out.n = prob->num_variables;
    out.m = prob->num_constraints;
    out.x = (double*)safe_malloc((size_t)out.n * sizeof(double));
    out.y = (double*)safe_malloc((size_t)out.m * sizeof(double));

    // copy solution from GPU to CPU
    hipError_t e1 = hipMemcpy(out.x, state->pdhg_primal_solution,
                                (size_t)out.n * sizeof(double), hipMemcpyDeviceToHost);
    hipError_t e2 = hipMemcpy(out.y, state->pdhg_dual_solution,
                                (size_t)out.m * sizeof(double), hipMemcpyDeviceToHost);
    if (e1 != hipSuccess || e2 != hipSuccess) {
        fprintf(stderr, "[interface] hipMemcpy failed: %s / %s\n",
                hipGetErrorName(e1), hipGetErrorName(e2));
        free(out.x); out.x = NULL;
        free(out.y); out.y = NULL;
        pdhg_solver_state_free(state);
        return out;
    }

    out.primal_obj = state->primal_objective_value;
    out.dual_obj = state->dual_objective_value;
    out.reason = state->termination_reason;

    pdhg_solver_state_free(state);
    return out;
}

// free lp solution
void lp_solution_free(lp_solution_t* sol) {
    if (!sol) return;
    if (sol->x) free(sol->x);
    if (sol->y) free(sol->y);
    sol->n = 0;
    sol->m = 0;
    sol->primal_obj = 0.0;
    sol->dual_obj   = 0.0;
    sol->reason = TERMINATION_REASON_UNSPECIFIED;
}